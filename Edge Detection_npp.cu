/*********************** Edge Detection using NPP Library functions ************************/
/***********************Filtering an Image using the vertical and horizonatal Prewitt kernels
/*********************** Name: Dinesh Kumar Sundararajan ******************************/ 
/*********************** UFID:61314525 ****************************************/
/*********************** Email : dsundar@ufl.edu **************************/
/******The file "result.txt" displays the parallel computation time 
**************************/

#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<npp.h>

double timerval()	//function to estimate time
{
	struct timeval st;
	gettimeofday(&st, NULL);
	return (st.tv_sec+st.tv_usec*1e-6);
}

int main()   //main function
{

    int i,j,k,n,count; //declaring variables

    for(i=3;i<20;i++)//loop for changing the input data size
    {
	double start = 0; //variables for time estimation
	double stop = 0;
	n = 2^i;//to change the data with every iteration of the loop
        count = n*n;
        
        float *src = (float *)malloc(count* sizeof(float)); // to initialize memory in CPU (source)
	float *dst = (float *)malloc(count* sizeof(float)); // to initialize memory in CPU (destination)

        for(k=0;k<count;k++)//loop to allocate random value for input image
	{
		src[k]=(rand()/255);
	}

        int gpu_n; //variable used to denote the number of data copied from CPU to GPU
  
        float *pSrc; //pSrc is the pointer to source image in GPU..
        float *pDst; //pDst is the pointer to destination image in GPU..

        hipMalloc(&pSrc, count * sizeof(float)); //to allocate memory for the input image in GPU
	hipMalloc(&pDst, count * sizeof(float)); //to allocate memory for the processed image in GPU
        
        hipMemcpy(pSrc, src, count * sizeof(float), hipMemcpyHostToDevice);//to copy the source image from CPU to GPU
	hipMemcpy(gpu_n, n, sizeof(int), hipMemcpyHostToDevice);//to copy the number of data to be processed from CPU to GPU 
        
        NppiSize oSizeROI = {n,n};

        float nSrcStep = n * sizeof(float);//to declare the size of the input image
	float nDstStep = n * sizeof(float);//to declare the size of the output image

        start=timerval();//start the timer

        for (j=0;j<1000;j++)//running the code 1000 times 
	{
            
                NppStatus nppiFilterPrewittHoriz_32f_C4R (const Npp32f *pSrc, Npp32s nSrcStep, Npp32f *pDst,Npp32s nDstStep, NppiSize oSizeROI); //Horizontal Prewitt Filter

                NppStatus nppiFilterPrewittVert_32f_C4R (const Npp32f *pSrc, Npp32s nSrcStep, Npp32f *pDst,Npp32s nDstStep, NppiSize oSizeROI); //Vertical Prewitt Filter
            
        }            

        stop=timerval();//stop the timer
	freopen("result.txt","a",stdout);//to store the computation time in result.txt
	printf("\n The computation time for %d * %d input size is : %f",n,n,((stop-start)/1000)); //to display the computation time
    
        hipFree(pSrc);
	hipFree(pDst);
	free(src);
	free(dst);


    }
}