      /*********************** Median filter using Array fire library for GPU **************/
/*********************** Ambareesh Chellappa UFID:1678 3293 Email : ambareesh.c@ufl.edu ***********/
/*create a file named result.txt in directory where the code is to be executed to store the results in the file */

#include<arrayfire.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
double timerval()	//function to obtain time
{
	struct timeval st;
	gettimeofday(&st, NULL);
	return (st.tv_sec+st.tv_usec*1e-6);
}
int main()
{
int i, x; //variable for the for loop
for(i=2;i<10;i++)//for loop to the change the data size
{
	double start = 0; // variable to calculate the time
	double stop = 0;
	x=2^i;//changing the data size every loop
	int count = x*x; 
	float *src = (float *)malloc(count* sizeof(float)); // initializing memory in CPU for source
	float *dest = (float *)malloc(count* sizeof(float)); // initializing memory in CPU for destination
	int k;
	for(k=0;k<count;k++)//loop to randomly allocate value for the input image
	{
		src[k]=(rand()/255);
	}
	float *gpu_src; //pointer for source img in GPU
	af_array inp, out; //array fire library declaration of array to process the data	
	hipMalloc(&gpu_src, count * sizeof(float)); //allocating memory for the input image to be processed in GPU
	hipMemcpy(gpu_src, src, count * sizeof(float), hipMemcpyHostToDevice);//copying source image from CPU to GPU
	array inp = array (x,x,src);//storing the input image in the form of array to be processed
	int j;
	start=timerval();
	for (j=0;j<1000;j++)//running the code 1000 times to avoid delays
	{
		//NppStatus nppiFilterMedian_32f_C4R (const Npp32f ∗pSrc, Npp32s nSrcStep, Npp32f ∗pDst,Npp32s nDstStep, NppiSize oSizeROI, NppiSize oMaskSize, NppiPoint oAnchor, Npp8u∗pBuffer)
		array out = af::medfilt(inp, 3, 3, AF_ZERO);//median filtering the provided input image
	}
	stop=timerval();
	freopen("result.txt","a",stdout);
	printf("\n the computation time for %d * %d input size is : %f",x,x,((stop-start)/1000)); /*displaying the computation time */
	hipFree(gpu_src);
	free(src);
	free(dest);
	}
}

	
