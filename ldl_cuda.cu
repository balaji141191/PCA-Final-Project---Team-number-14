#include "hip/hip_runtime.h"
/*********************** LDL Decomposition using CUDA ************************/
/*********************** Name: Balaji Rajasekaran ******************************/ 
/*********************** UFID:1918-2684 ****************************************/
/*********************** Email : balaji141191@ufl.edu **************************/
/*********************** Team-number : 14***************************************/


#include<stdio.h>
#include<stdlib.h>
#include<assert.h>
#include<math.h>
#include<time.h>

#include<hipsolver.h>
#include <hipblas.h>

#include<sys/time.h>
#define EPISILON (0.0001)
#define MAX_VALUE (1e6)

//function to compute the execution time
double timerval()	
{
	struct timeval st;
	gettimeofday(&st, NULL);
	return (st.tv_sec+st.tv_usec*1e-6);
}

//function to generate a positive definite matrix
void init(int n, double* A, double* B) 
{
	int i, j, k;
	
	//creating a random matrix
	for(i=0; i<n; i++) 
	{
		for(j=i; j<n; j++) 
		{
			double r = rand() % 100;
			A[i*n+j] = r;
			A[j*n+i] = A[i*n+j];
		}
	}

	
	//converting to positive definite matrix
	for(i=0; i<n; i++) 
	{
		for(j=0; j<n; j++) 
		{
			double s = 0;
			for(k=0; k<n; k++) 
			{
				s += A[i*n+k]*A[j*n+k];
			}
			B[i*n+j] = s;
		}
	}
}


//Main function
int main()
{
	double startTime = 0;
   	double endTime = 0;
   	int i, j, n;
	int info;
	    	
	hipsolverStatus_t buffer_Status, solve_status;
	
	
	for(i=1; i<13; i++) 
	{
		n = pow(2, i);	//Run the routine for matrix of order 2 to 4096
	
		double *A = (double *)malloc(n* n * sizeof(double));
		double *B = (double *)malloc(n* n * sizeof(double));
		
		init(n, A, B); //initialize matrix function call
		
		double *M;
		hipMalloc(&M, n* n * sizeof(double));		//Allocate memory for M in GPU

		hipMemcpy(M, B, n * n * sizeof(double), hipMemcpyHostToDevice);	//Copy contents of the initialised array from host to device memory
		
		int *devInfo; 
		hipMalloc(&devInfo, sizeof(int));

		hipsolverHandle_t handle;	//Initializing the CUDA solver	
    		hipsolverDnCreate(&handle);
		
		int work_size = 0;
		buffer_status = hipsolverDnDsytrf_bufferSize(handle, n, M, n, &work_size));	//CUDA sytrf initialization
		
		double *work;
		hipMalloc(&work, work_size * sizeof(double));
		int I[n];
		
		startTime = timerval();
		for(j=0; j<1000; j++)	//Running the code 1000 times
		{
			
			solve_status = hipsolverDnDsytrf(handle, HIPBLAS_FILL_MODE_LOWER, n, M, n, I, work, work_size, devInfo);	//CUDA sytrf function execution
						
		} 
		endTime = timerval();
		freopen("ldl_CUDA_results.txt","a",stdout);
		printf("\n The computation time for %d order matrix is : %f \n", n, ((endTime - startTime)/1000)); //Print the execution time 

		hipsolverDnDestroy(handle);
		
		hipFree(M);
		free(B);
		free(A);		
	}
return 0;
}

