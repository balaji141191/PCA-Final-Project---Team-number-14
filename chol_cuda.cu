#include "hip/hip_runtime.h"
/*-------------------------------- KERNEL : CHOLESKY DECOMPOSITION (USING cuSolver [CUDA] LIBRARY) --------------------*/
/*-------------------------------- TEAM : 14 --------------------------------------------------------------------------*/
/*-------------------------------- NAME : ATHIRA AJAYAKUMAR -----------------------------------------------------------*/
/*-------------------------------- UFID : 69398411 --------------------------------------------------------------------*/
/*-------------------------------- EMAIL ID : athira010192@ufl.edu ----------------------------------------------------*/ 


//header file declaration
#include<stdio.h>
#include<stdlib.h>
#include<assert.h>
#include<math.h>
#include<hipsolver.h>
#include <hipblas.h>
#include<time.h>
#include<sys/time.h>
#define EPISILON (0.0001)
#define MAX_VALUE (1e6)


//function to compute the execution time
double timerval()	
{
	struct timeval st;
	gettimeofday(&st, NULL);
	return (st.tv_sec+st.tv_usec*1e-6);
}



//function to initialize the matrix as a positive definite matrix
void initialize(int n, double* A, double* P)	
{
	int i, j, k;
	
	//generating a random matrix
	for(i=0; i<n; i++) 
	{
		for(j=i; j<n; j++) 
		{
			double r = rand() % 100;
			A[i*n+j] = r;
			A[j*n+i] = A[i*n+j];
		}
	}

	
	//converting to positive definite matrix
	for(i=0; i<n; i++) 
	{
		for(j=0; j<n; j++) 
		{
			double sum = 0;
			for(k=0; k<n; k++) 
			{
				sum += A[i*n+k]*A[j*n+k];
			}
			P[i*n+j] = sum;
		}
	}
}



//main function
int main()
{
	double startTime = 0;
   	double endTime = 0;
   	int i, j, n;
	hipsolverStatus_t b_Status, s_status;
		
	for(i=1; i<13; i++) 
	{
		n = pow(2, i);
		
		//allocating memory for CPU variables
		double *A = (double *)malloc(n* n * sizeof(double));
		double *P = (double *)malloc(n* n * sizeof(double));
		initialize(n, A, P); //function call to initialize matrix
		
		//allocating memory for GPU variables
		double *B;
		hipMalloc(&B, n* n * sizeof(double));
		hipMemcpy(B, P, n * n * sizeof(double), hipMemcpyHostToDevice);
		int *devInfo; 
		hipMalloc(&devInfo, sizeof(int));
		int work_size = 0;

		//cuSolver initialization functions
		hipsolverHandle_t handle;
    		hipsolverDnCreate(&handle);
		
		b_status = hipsolverDnDpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_LOWER, n, B, n, &work_size); //function for allocating buffer size
		
		double *work;
		hipMalloc(&work, work_size * sizeof(double));
				
		startTime = timerval();
		for(j=0; j<1000; j++)	//Running the code 1000 times
		{
			
			s_status = hipsolverDnDpotrf(handle, HIPBLAS_FILL_MODE_LOWER, n, B, n, work, work_size, devInfo); //function for performing cholesky decomposition
						
		} 
		endTime = timerval();
		freopen("cholesky.txt","a",stdout);
		printf("\n The computation time for %d order matrix is : %f \n", n, ((endTime - startTime)/1000)); //Print the execution time 

		hipsolverDnDestroy(handle);		
		hipFree(B);
		free(P);
		free(A);		
	}
	return 0;
}
