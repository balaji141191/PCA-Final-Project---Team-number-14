/*********************** Edge Detection using ArrayFire Library functions ************************/
/***********************Using the Sobel filter*****************************************/
/*********************** Name: Dinesh Kumar Sundararajan ******************************/ 
/*********************** UFID:61314525 ****************************************/
/*********************** Email : dsundar@ufl.edu **************************/
/******The file "result.txt" displays the parallel computation time 
**************************/

#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<arrayfire.h>

double timerval()	//function to estimate time
{
	struct timeval st;
	gettimeofday(&st, NULL);
	return (st.tv_sec+st.tv_usec*1e-6);
}

int main()   //main function
{

    int i,j,k,n,count; //declaring variables

    for(i=3;i<20;i++)//loop for changing the input data size
    {
	double start = 0; //variables for time estimation
	double stop = 0;
	n = 2^i;//to change the data with every iteration of the loop
        count = n*n;
        
        float *src = (float *)malloc(count* sizeof(float)); // to initialize memory in CPU (source)
	float *dst = (float *)malloc(count* sizeof(float)); // to initialize memory in CPU (destination)

        for(k=0;k<count;k++)//loop to allocate random value for input image
	{
		src[k]=(rand()/255);
	}
        
         
        af_array img, output; //array fire library declaration of array to process the data
        af_array img = array (n,n,src);//storing the input image in the form of array to be processed
     
       
        start=timerval();//start the timer

        for (j=0;j<1000;j++)//running the code 1000 times 
	{
             af_array output = AFAPI af_err af_sobel_operator( af_array *dx, af_array *dy, const af_array img, const unsigned ker_size = 3); //library function to perform edge detection       
            
        }            

        stop=timerval();//stop the timer
	freopen("result.txt","a",stdout);//to store the computation time in result.txt
	printf("\n The computation time for %d * %d input size is : %f",n,n,((stop-start)/1000)); //to display the computation time
    
        hipFree(img);
	hipFree(output);
	free(src);
	free(dst);

    }
}