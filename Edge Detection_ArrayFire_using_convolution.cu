#include "hip/hip_runtime.h"
/*********************** Edge Detection using ArrayFire Library functions ************************/
/***********************Using Convolution and gradient functions for Prewitt Operator*****************************************/
/*********************** Name: Dinesh Kumar Sundararajan ******************************/ 
/*********************** UFID:61314525 ****************************************/
/*********************** Email : dsundar@ufl.edu **************************/
/******The file "result.txt" displays the parallel computation time 
**************************/

#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<arrayfire.h>

double timerval()	//function to estimate time
{
	struct timeval st;
	gettimeofday(&st, NULL);
	return (st.tv_sec+st.tv_usec*1e-6);
}

int main()   //main function
{

    int i,j,k,n,count; //declaring variables
    //kernels for prewitt operator
    float h1[] = { 1, 1, 1};
    float h2[] = {-1, 0, 1};
 
    for(i=3;i<20;i++)//loop for changing the input data size
    {
	double start = 0; //variables for time estimation
	double stop = 0;
	n = 2^i;//to change the data with every iteration of the loop
        count = n*n;
        
        float *src = (float *)malloc(count* sizeof(float)); // to initialize memory in CPU (source)
	float *dst = (float *)malloc(count* sizeof(float)); // to initialize memory in CPU (destination)

        for(k=0;k<count;k++)//loop to allocate random value for input image
	{
		src[k]=(rand()/255);
	}
        
         
        array dir, mag; //array fire library declaration to compute magnitude and direction
        const array in = array (n,n,src);//storing the input image in the form of array to be processed
       
        start=timerval();//start the timer

        for (j=0;j<1000;j++)//running the code 1000 times 
	{
            
         // Finding the gradients
         array Gy = convolve(3, h2, 3, h1, in)/6;
         array Gx = convolve(3, h1, 3, h2, in)/6;
 
         // Find magnitude and direction
         mag = hypot(Gx, Gy);
         dir = atan2(Gy, Gx);

        }            

        stop=timerval();//stop the timer
	freopen("result.txt","a",stdout);//to store the computation time in result.txt
	printf("\n The computation time for %d * %d input size is : %f",n,n,((stop-start)/1000)); //to display the computation time
    
        hipFree(in);
	hipFree(Gx);
        hipFree(Gy);
        hipFree(mag);
        hipFree(dir);
	free(src);
	free(dst);

    }
}