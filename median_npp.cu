      /*********************** Median filter using NPP library for GPU **************/
/*********************** Ambareesh Chellappa UFID:1678 3293 Email : ambareesh.c@ufl.edu ***********/
/*create a file named result.txt in directory where the code is to be executed to store the results in the file */

#include <npp.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
double timerval()	//function to obtain time
{
	struct timeval st;
	gettimeofday(&st, NULL);
	return (st.tv_sec+st.tv_usec*1e-6);
}
int main()
{
int i, x; //variable for the for loop
for(i=2;i<10;i++)//for loop to the change the data size
{
	double start = 0; // variable to calculate the time
	double stop = 0;
	x=2^i;//changing the data size every loop
	int count = x*x; 
	float *src = (float *)malloc(count* sizeof(float)); // initializing memory in CPU for source
	float *dest = (float *)malloc(count* sizeof(float)); // initializing memory in CPU for destination
	int k;
	for(k=0;k<count;k++)//loop to randomly allocate value for the input image
	{
		src[k]=(rand()/255);
	}
	int gpu_x;
	float *gpu_src, *gpu_dest; //pointer for source img in GPU
	int *gpu_buffer; 	
	hipMalloc(&gpu_src, count * sizeof(float)); //allocating memory for the input image to be processed in GPU
	hipMalloc(&gpu_dest, count * sizeof(float)); //allocating memory for the processed image in GPU
	hipMalloc(&gpu_buffer, count * sizeof(int)); //allocating memory for the processed image in GPU
	hipMemcpy(gpu_src, src, count * sizeof(float), hipMemcpyHostToDevice);//copying source image from CPU to GPU
	hipMemcpy(gpu_x, x, sizeof(int), hipMemcpyHostToDevice);//copying number of data to be processed from CPU to GPU
	NppiSize oMaskSize = {3, 3};
    NppiSize oSizeROI = {x,x};
    NppiPoint oAnchor = {0, 0};
	float nSrcStep = x * sizeof(float);//declaring the size of the input image
	float nDstStep = x * sizeof(float);//declaring the size of the output image
    int j;
	start=timerval();
	for (j=0;j<1000;j++)//running the code 1000 times to avoid delays
	{
		NppStatus nppiFilterMedian_32f_C4R (const Npp32f *gpu_src, Npp32s nSrcStep, Npp32f *gpu_dest, Npp32s nDstStep, NppiSize oSizeROI, NppiSize oMaskSize, NppiPoint oAnchor, Npp8u *gpu_buffer)
	}
	stop=timerval();
	freopen("result.txt","a",stdout);
	printf("\n the computation time for %d * %d input size is : %f",x,x,((stop-start)/1000)); /*displaying the computation time */
	hipFree(gpu_src);
	hipFree(gpu_dest);
	hipFree(gpu_buffer);
	free(src);
	free(dest);
	}
}

	
